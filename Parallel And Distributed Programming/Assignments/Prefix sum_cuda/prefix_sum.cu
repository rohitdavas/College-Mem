#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cmath>

#define POWER 24
#define THREAD 1024 

using namespace std;

__global__
void add_kernel(double *d_a, double *d_tmp, long k, long n)
{
    long i = blockIdx.x*blockDim.x + threadIdx.x;

    if (i + k < n)
    {
        d_tmp[i+k] = d_a[i+k] + d_a[i];
    }
}


__host__
double verify(double *a, double *b, long n)
{
    double *v; 
    v = (double *) malloc(n*sizeof(double));

    for (int i = 0; i < n; i++)
    {
        v[i] = a[i];
    }

    for (int i = 1; i<n; i++)
    {
        v[i] = v[i] + v[i-1];
    }

    for (int i = 1; i < n; i++)
    {
        v[i] = v[i] + v[i-1];
    }

    double maxError = 0; 
    for (int i = 0; i < n; i++)
    {
        maxError = fmax(maxError, fabs(v[i] - b[i]));
    }

    return maxError/v[n-1];
}



int main(int argc, const char *argv[])
{       
    if(argc == 1)
    {
        printf("filling the file with random inputs..\n");

        FILE *fp = fopen("./input","w");

        if (fp == NULL){
            printf("file pointer cannot be created\n");
            return 1; 
        }

        int n = 1 << 24;
        printf("no of elemets are: %d\n",n);
        fprintf(fp, "%d\n", n);

        srand(time(NULL));

        for (int i=0; i<n; i++)
            fprintf(fp, "%lg\n", ((double)(rand() % n))/100);

        fclose(fp);
        printf("Finished writing\n");
    }

    else if(argc ==2)
    {
        printf("input file provided. data will be read from it.");
    }


    FILE *fp = fopen("input","r");

    if (fp == NULL)
    {
        printf("there must be a file created in your current folder. please check and update the fopen.");
        return 1;
    }

    long n;
    fscanf(fp, "%ld\n", &n);
    printf("value of n: %ld\n",n);

    double *a = (double *)malloc(n*sizeof(double));
    double *b = (double *)malloc(n*sizeof(double));

    printf("reading the file... ");
    for(int i = 0; i<n; i++)
    {
        fscanf(fp, "%lg\n", &a[i]);
    } 
    printf("done.\n");

    fclose(fp);  

    printf("gpu computation start ...\n");
    //allocate memory on gpu 
    double *d_a, *d_tmp;
    hipMalloc(&d_a, n*sizeof(double));
    hipMalloc(&d_tmp, n*sizeof(double)); 

    // copy from cpu to gpu 
    hipMemcpy(d_a, a, n*sizeof(double), hipMemcpyHostToDevice);

    // copy content into temporary array
    hipMemcpy(d_tmp, d_a, n*sizeof(double), hipMemcpyDeviceToDevice);

    // first pass 

    for(long p = 0; p<= POWER; p++){
        add_kernel << <(n+THREAD -1)/THREAD, THREAD>>>(d_a, d_tmp,1<<p, n);
        hipMemcpy(d_a, d_tmp, n*sizeof(double), hipMemcpyDeviceToDevice);
    }


    //second pass 
    for(long p = 0; p<= POWER; p++){
        add_kernel << <(n+THREAD -1)/THREAD, THREAD>>>(d_a, d_tmp,1<<p, n);
        hipMemcpy(d_a, d_tmp, n*sizeof(double), hipMemcpyDeviceToDevice);
    }

    //copy back to cpu 
    hipMemcpy(b, d_a, n*sizeof(double), hipMemcpyDeviceToHost);

    //free memory on gpu 
    hipFree(d_a);
    hipFree(d_tmp);

    printf("gpu computation done.\n");

    //verify the answer
    printf("verifying results of GPU with classical approach\n");
    double max_error = verify(a, b, n);
    printf("error margin: %f\n\n", max_error);

}