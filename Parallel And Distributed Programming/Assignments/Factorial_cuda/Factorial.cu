// #include <LEDA/numbers/integer.h>

#include <hip/hip_runtime.h>
#include<stdio.h>



__global__ void Factorial(int *gpu_num,long int *gpu_res)
{
  int i;
  *gpu_res=1;
  for(i=1;i<=*gpu_num;i++)
  {
    *gpu_res = *gpu_res * i;      
  }
}

int main()
{
  int Number;  //to store number on the cpu/host machine
  int *dev_number;
  long int *res; //store result 
  unsigned long long int result;
  system("clear"); //to clear the screen
  printf("\n\t Enter the number : ");
  scanf("%d",&Number);
  
  //to allocate memory for a number on the GPU/Device
  hipMalloc((void**)&dev_number,sizeof(int));
  hipMalloc((void**)&res,sizeof(unsigned long long int));
  
  //copy number to the GPU/Device memory
  hipMemcpy(dev_number,&Number,sizeof(int),hipMemcpyHostToDevice);
 
  //call square function which will execute parallely on GPU
  Factorial<<<1,1000>>>(dev_number,res);

  //copy result back from device/GPU back to CPU/Host
  hipMemcpy(&result,res,sizeof(long int),hipMemcpyDeviceToHost);

  //display result on the screen
  printf("\n\t Factorial of number %d is %lld \n",Number,result); 
 
  //deallocate GPU/Device memory
  return 0; 
}

